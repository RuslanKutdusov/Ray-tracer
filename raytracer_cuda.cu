#include "hip/hip_runtime.h"
#include "Vector.hpp"
#include "Ray.hpp"
#include "Material.hpp"
#include "Texture.hpp"
#include <stdint.h>
#include <stdio.h>

#define EPSILON 0.0001f
#define PI 3.1415926f

__constant__ Material 		g_materials[100];
__constant__ uint32_t 		g_materialsNumber;


struct Intersection
{
    Vector  point;
    Vector  normal;
    float	tx;
    float 	ty;
    float 	padding[ 2 ];
} __attribute__ ( ( aligned( 16 ) ) );


//
HOST_DEVICE void GetReflectRefractVectors( const Material& material, const Ray & ray, const Intersection& intersection, Vector& reflect, Vector& refract, float& reflectAmount )
{
    Vector i = ray.vector;
    Vector n = intersection.normal;
    reflect = i.reflect( n );
    reflect.normalize();
    reflectAmount = 1.0f;
    if ( material.m_refract_amount > 0 )
    {
        float refract_coef = material.m_refract_coef;
        float cos_i = -i.dot( n );
        if( cos_i < 0.0f )
        {
            n = n.scalar( -1 );
            cos_i = -i.dot( n );
            refract_coef = 1.0f / refract_coef;
        }
        float sin2_t = refract_coef * refract_coef * ( 1.0f - cos_i * cos_i );

        if( sin2_t <= 1.0f  )
        {
            float cos_t = sqrt( 1.0f - sin2_t );
            refract = i.scalar( refract_coef ) + n.scalar( refract_coef * cos_i - cos_t );
            refract.normalize();
            float Rorto  = ( cos_i - refract_coef * cos_t ) / ( cos_i + refract_coef * cos_t );
            float Rparal = ( refract_coef * cos_i - cos_t ) / ( refract_coef * cos_i + cos_t );
            reflectAmount = ( Rorto * Rorto + Rparal * Rparal ) / 2.0f;
        }
        else
        {
            refract = reflect;
        }
    }
}


//
HOST_DEVICE bool is_in_border( const float & v, const float & b1, const float & b2, float & t )
{
	t = ( v - b1 ) / ( b2 - b1 );
	t = t > 1.0f ? 1.0f : t;
	if( ( v >= b1 && v <= b2 ) )// || ( v >= b2 && v <= b1 ) )
			return true;
	if( fabs( v - b1 ) < EPSILON || fabs( v - b2 ) < EPSILON )
		return true;
	return false;
}


//
struct ObjectPlane
{
	uint32_t 	material;
	Vector 		abcd;
	Vector 		normal;
	Vector 		b1;
	Vector 		b4;
	Matrix 		inverse;
	HOST_DEVICE ObjectPlane()
	{

	}
	HOST_DEVICE ObjectPlane( const Matrix & m, float width, float height,
							uint32_t material_, bool inverse_normal = false )
	{
		inverse = m.inverse();
		b1 = Vector( -width / 2.0f, height / 2.0f, 0.0f );
		Vector b2 = Vector( width / 2.0f, height / 2.0f, 0.0f );
		b4 = Vector( width / 2.0f, -height / 2.0f, 0.0f );
		Vector c = m.mul( Vector( 0.0f, 0.0f, 0.0f ) );
		Vector u = m.mul( b1 ) - c;
		Vector v = m.mul( b2 ) - c;
		normal = ( u * v );
		if( !inverse_normal )
			normal = normal.scalar( -1 );
		normal.normalize();
		abcd = Vector( normal.x, normal.y, normal.z, -normal.dot( c ) );
		material = material_;
	}
	HOST_DEVICE bool CheckIntersection( const Ray& ray, Intersection & intersection )
	{
		intersection.normal = normal;
		const float& A = abcd.x;
		const float& B = abcd.y;
		const float& C = abcd.z;
		const float& D = abcd.w;
		const float& x0 = ray.start_point.x;
		const float& y0 = ray.start_point.y;
		const float& z0 = ray.start_point.z;
		const float& alfa =  ray.vector.x;
		const float& beta = ray.vector.y;
		const float& gamma = ray.vector.z;
		//( A,B,C ) -normal
		//( a,b,g ) - ray direction
		float scalar = A * alfa + B * beta + C * gamma;
		//прамая || плоскости
		if( fabs( scalar ) < EPSILON )
			return false;
		float t = ( -D - A * x0 - B * y0 - C * z0 ) / scalar;
		//точка должна быть по направлению луча
		if( t < 0 || fabs( t ) < EPSILON )
			return false;
		intersection.point = ray.point( t );

		Vector intr = inverse.mul( intersection.point );
		if ( !is_in_border( intr.x, b1.x, b4.x, intersection.tx ) || !is_in_border( intr.y, b4.y, b1.y, intersection.ty ) )
			return false;

		return true;
	}
};

struct ObjectSphere
{
public:
	uint32_t 	material;
	Vector  	position;
    float  		radius;

    HOST_DEVICE ObjectSphere() 
    {

    }

    HOST_DEVICE ObjectSphere( const Vector & position_, float radius_, uint32_t material_ )
       : position( position_ ), radius( radius_ ), material( material_ )
    {

    }

    HOST_DEVICE bool intersect( const Ray &ray, const float & t, Intersection & intersection ) const
    {
        intersection.point = ray.point( t );
        if( fabs( ( intersection.point - position ).length() - radius ) > EPSILON )
               return false;

        intersection.normal = intersection.point - position;
        intersection.normal.normalize();
        intersection.tx = 0.0f;
        intersection.ty = 0.0f;

        return true;
    }

    HOST_DEVICE bool CheckIntersection( const Ray &ray, Intersection & intersection ) const
    {
        const float & R = radius;
        Vector v = ray.start_point - position;
        float B = v.dot( ray.vector );
        float C = v.dot( v ) - R * R;
        float D = B * B - C;
        if( D < 0.0f )
        	return false;
        D = sqrtf( D );
        float t1 = ( -B - D );
        float t2 = ( -B + D );
        if( t1 < 0.0f && t2 < 0.0f )
            return false;
        float min_t = fmin( t1, t2 );
        float max_t = fmax( t1, t2 );
        float t = ( min_t >= 0 ) ? min_t : max_t;
        if( fabs( t ) < EPSILON )
        {
            if ( t < EPSILON )
                t = max_t;
            if ( t < EPSILON )
                return false;
        }
        return intersect( ray, t, intersection );
    }
};

struct PointLight
{
	Color		color;
	Vector 		position;
	float		radius;

	HOST_DEVICE PointLight()
	{

	}

	HOST_DEVICE PointLight( const Color & c, const Vector & pos, float radius_ )
	{
		color = c;
		position = pos;
		radius = radius_;
	}
	
	PointLight operator=( const PointLight& pl )
	{
		color = pl.color;
		position = pl.position;
		radius = pl.radius;
		return *this;
	}

	HOST_DEVICE float distance( const Vector & point ) const
	{
		return position.distance( point );
	}
};

__constant__ PointLight 	g_pointLights[100];
__constant__ uint32_t 		g_pointLightsNumber;
__constant__ ObjectPlane 	g_objPlanes[100];
__constant__ uint32_t 		g_objPlanesNumber;
__constant__ ObjectSphere 	g_objSpheres[100];
__constant__ uint32_t 		g_objSpheresNumber;


//
__device__ bool CheckIntersection( uint32_t objectIndex, const Ray & ray, Intersection & intr )
{
	if( objectIndex < g_objPlanesNumber )
		return g_objPlanes[ objectIndex ].CheckIntersection( ray, intr );
	const uint32_t& nextIndex = objectIndex - g_objPlanesNumber;
	if( nextIndex < g_objSpheresNumber )
		return g_objSpheres[ nextIndex ].CheckIntersection( ray, intr );
	return false;
}


//
__device__ const Material& GetMaterial( uint32_t objectIndex )
{
	if( objectIndex < g_objPlanesNumber )
		return g_materials[ g_objPlanes[ objectIndex ].material ];
	const uint32_t& nextIndex = objectIndex - g_objPlanesNumber;
	if( nextIndex < g_objSpheresNumber )
		return g_materials[ g_objSpheres[ nextIndex ].material ];
	return g_materials[ nextIndex ];
}


//
__device__ Color ray_trace( const Ray & ray, Intersection & intr, uint32_t objNumber )
{
	uint32_t i_object = ~0u;
	float distance2obj = INFINITY;

	for( uint32_t i = 0; i < objNumber; i++ )
	{
		Intersection in;
		if( CheckIntersection( i, ray, in ) )
		{
			const float& dist = in.point.distance( ray.start_point );
			if( dist < distance2obj )
			{
				distance2obj = dist;
				intr = in;
				i_object = i;
			}
		}
	}
	if( i_object == ~0u )
		return Color( 0.0f, 0.0f, 0.0f );

	Ray reflectRay;
	Ray refractRay;
	float reflectAmount;

	const Material& material = GetMaterial( i_object );
	Color ret = material.m_ambient;

	reflectRay.start_point = intr.point;
	refractRay.start_point = intr.point;
	GetReflectRefractVectors( material, ray, intr, reflectRay.vector, refractRay.vector, reflectAmount );

	for( uint32_t i = 0; i < g_pointLightsNumber; i++ )
	{
		const float distance2light = g_pointLights[ i ].distance( intr.point );
		Vector fromLight = intr.point - g_pointLights[ i ].position;
		Ray to_light( g_pointLights[ i ].position, intr.point );

		//проверям, в тени какого либо объекта или нет
		bool i_object_in_shadow = false;
		for( uint32_t j = 0; j < objNumber; j++ )
		{
			Intersection intr2;
			if( CheckIntersection( j, to_light, intr2 ) )
			{
				const float& distance_ = intr2.point.distance( intr.point );
				if( distance_ < distance2light )
				{
					i_object_in_shadow = true;
					break;
				}
			}
		}
		if( i_object_in_shadow )
			continue;

		float attenuation = distance2light / g_pointLights[ i ].radius;
		if( attenuation > 1.0f )
			continue;
		attenuation = 1.0f - attenuation;

		float angle_cos = to_light.vector.dot( intr.normal );
		if( angle_cos > 0 )
			if( !material.m_diffuse.is_black() )
				ret = ret + g_pointLights[ i ].color * angle_cos * material.m_diffuse * attenuation;

		angle_cos = to_light.vector.dot( reflectRay.vector );
		if( angle_cos > 0 )
			if( !material.m_specular.is_black() )
				ret = ret + g_pointLights[ i ].color * pow( angle_cos, material.m_phong ) * attenuation;
	}

	return ret;
}

__global__ void calculate_light( Color* image, uint32_t width, uint32_t height, Vector cameraPos, Viewport viewport, uint32_t objNumber )
{
	uint32_t pixel_index = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t yi = pixel_index / width;
	uint32_t xi = pixel_index - yi * width;
	float delta_x = ( viewport.m_p2.x - viewport.m_p1.x ) / ( float )width;
	float delta_z = ( viewport.m_p2.z - viewport.m_p1.z ) / ( float )width;
	float delta_y = ( viewport.m_p1.y - viewport.m_p3.y ) / ( float )height;
	float x = delta_x * xi + viewport.m_p1.x;
	float y = viewport.m_p1.y - delta_y * yi;
	float z = delta_z * xi + viewport.m_p1.z;

	Ray ray( Vector( x, y, z ), cameraPos );
	Intersection intersection;

	uint32_t i_object = ~0u;
	float distance2obj = INFINITY;

	for( uint32_t i = 0; i < objNumber; i++ )
	{
		Intersection in;
		if( CheckIntersection( i, ray, in ) )
		{
			const float& dist = in.point.distance( ray.start_point );
			if( dist < distance2obj )
			{
				distance2obj = dist;
				intersection = in;
				i_object = i;
			}
		}
	}
	if( i_object == ~0u )
		return Color( 0.0f, 0.0f, 0.0f );

	Ray reflectRay;
	Ray refractRay;
	float reflectAmount;

	const Material& material = GetMaterial( i_object );
	Color ret = material.m_ambient;

	reflectRay.start_point = intersection.point;
	refractRay.start_point = intersection.point;
	GetReflectRefractVectors( material, ray, intersection, reflectRay.vector, refractRay.vector, reflectAmount );

	for( uint32_t i = 0; i < g_pointLightsNumber; i++ )
	{
		const float distance2light = g_pointLights[ i ].distance( intersection.point );
		Vector fromLight = intersection.point - g_pointLights[ i ].position;
		Ray to_light( g_pointLights[ i ].position, intersection.point );

		//проверям, в тени какого либо объекта или нет
		bool i_object_in_shadow = false;
		for( uint32_t j = 0; j < objNumber; j++ )
		{
			Intersection intr2;
			if( CheckIntersection( j, to_light, intr2 ) )
			{
				const float& distance_ = intr2.point.distance( intersection.point );
				if( distance_ < distance2light )
				{
					i_object_in_shadow = true;
					break;
				}
			}
		}
		if( i_object_in_shadow )
			continue;

		float attenuation = distance2light / g_pointLights[ i ].radius;
		if( attenuation > 1.0f )
			continue;
		attenuation = 1.0f - attenuation;

		float angle_cos = to_light.vector.dot( intersection.normal );
		if( angle_cos > 0 )
			if( !material.m_diffuse.is_black() )
				ret = ret + g_pointLights[ i ].color * angle_cos * material.m_diffuse * attenuation;

		angle_cos = to_light.vector.dot( reflectRay.vector );
		if( angle_cos > 0 )
			if( !material.m_specular.is_black() )
				ret = ret + g_pointLights[ i ].color * pow( angle_cos, material.m_phong ) * attenuation;
	}

	image[ pixel_index ] = ret;

//	float d = 0;

//	const float & R = intr.reflect_amount;
//	float T = 1.0 - R;
//
//	Color reflect_ray_color = ray_tracing( intr.reflect_ray, depth_, rays_count, &d );
//	reflect_ray_color = reflect_ray_color * exp( -objects[i_object]->m_material.m_beta ) * R;
//
//	Color refract_ray_color;
//	if ( objects[i_object]->m_material.m_refract_amount > 0 && T > EPSILON )
//		refract_ray_color = ray_tracing( intr.refract_ray, depth_, rays_count, NULL ) * objects[i_object]->m_material.m_refract_amount * T;
//
//	ret = objects[i_object]->m_material.m_ambient +
//		  objects[i_object]->m_material.m_diffuse * diffuse * intr.pixel +
//		  objects[i_object]->m_material.m_specular * specular +
//			reflect_ray_color +
//			refract_ray_color ;


//	image[ pixel_index ].r = 0;
//	image[ pixel_index ].g = 0;
//	image[ pixel_index ].b = 0;
//	if( fabs( x ) < 0.1f && y > 0.0f )
//		image[ pixel_index ].r = 1.0f;
//	if( fabs( x ) < 0.1f && y < 0.0f )
//		image[ pixel_index ].g = 1.0f;
//	if( fabs( y ) < 0.1f && x > 0.0f )
//		image[ pixel_index ].b = 1.0f;
//	if( fabs( y ) < 0.1f && x < 0.0f )
//	{
//		image[ pixel_index ].g = 1.0f;
//		image[ pixel_index ].b = 0.5f;
//	}
}

#define CUDA_CHECK_RETURN( value ) {											\
	hipError_t _m_cudaStat = value;										\
	if ( _m_cudaStat != hipSuccess ) {										\
		fprintf( stderr, "Error '%s' at line %d in file %s\n",					\
				hipGetErrorString( _m_cudaStat ), __LINE__, __FILE__ );		\
		exit( 1 );															\
	} }

int main()
{
	Material m1( Color( 0.01f, 0.01f, 0.01f ), Color( 0.8f, 0.8f, 0.8f ), Color( 0.5f, 0.5f, 0.7f ), 5.0f, 20.0f, 0.0f, 0.0f );
	uint32_t matNumber = 1;

	const uint32_t lightsNumber = 2;
	PointLight pointLights[ lightsNumber ];
	pointLights[ 0 ] = PointLight( Color( 1.0f, 0.0f, 0.0f ), Vector( -2.0f, 4.0f, -1.0f ), 10.0f );
	pointLights[ 1 ] = PointLight( Color( 0.0f, 1.0f, 0.0f ), Vector( 2.0f, 4.0f, -1.0f ), 10.0f );

	Matrix m = Matrix::RotateX( PI / 2.0f );
	m = m * Matrix::TranslateMatrix( 0.0f, -3.0f, 0.0f );
	ObjectPlane plane( m, 100.0f, 100.0f, 0, true );
	uint32_t planesNumber = 1;

	ObjectSphere sphere( Vector( 0.0f, 0.0f, 0.0f ), 1.5f, 0 );
	uint32_t spheresNumber = 1;

	CUDA_CHECK_RETURN( hipMemcpyToSymbol(HIP_SYMBOL( g_materials), &m1, sizeof( Material ), 0 ) );
	CUDA_CHECK_RETURN( hipMemcpyToSymbol(HIP_SYMBOL( g_materialsNumber), &matNumber, sizeof( uint32_t ) * matNumber, 0 ) );

	CUDA_CHECK_RETURN( hipMemcpyToSymbol(HIP_SYMBOL( g_pointLights), &pointLights, sizeof( PointLight ) * lightsNumber, 0 ) );
	CUDA_CHECK_RETURN( hipMemcpyToSymbol(HIP_SYMBOL( g_pointLightsNumber), &lightsNumber, sizeof( uint32_t ), 0 ) );

	CUDA_CHECK_RETURN( hipMemcpyToSymbol(HIP_SYMBOL( g_objPlanes), &plane, sizeof( ObjectPlane ) * planesNumber, 0 ) );
	CUDA_CHECK_RETURN( hipMemcpyToSymbol(HIP_SYMBOL( g_objPlanesNumber), &planesNumber, sizeof( uint32_t ), 0 ) );

	CUDA_CHECK_RETURN( hipMemcpyToSymbol(HIP_SYMBOL( g_objSpheres), &sphere, sizeof( ObjectSphere ) * spheresNumber, 0 ) );
	CUDA_CHECK_RETURN( hipMemcpyToSymbol(HIP_SYMBOL( g_objSpheresNumber), &spheresNumber, sizeof( uint32_t ), 0 ) );

	image_t image;
	image.width = 2048;
	image.height = 2048;
	uint32_t image_size = image.width * image.height;

	image.image = new Color[ image_size ];

	float aspectRatio = ( float )image.width / ( float )image.height;
	Vector cameraPos( 0.0f, 0.0f, -10.0f );
	float viewportWidth = 6.0f;
	float viewportHeight = viewportWidth / aspectRatio;
	float f = -5.0f;
	Viewport viewport( Vector( -viewportWidth / 2.0f,  viewportHeight / 2.0f, f ),
					  Vector( viewportWidth / 2.0f,  viewportHeight / 2.0f, f ),
					  Vector( -viewportWidth / 2.0f, -viewportHeight / 2.0f, f ),
					  Vector( viewportWidth / 2.0f, -viewportHeight / 2.0f, f ) );

	Color* image_device;
	CUDA_CHECK_RETURN( hipMalloc( &image_device, image_size * sizeof( Color ) ) );
//	ObjectPlane* objPlane_device;
//	CUDA_CHECK_RETURN( hipMalloc( &objPlane_device, 1 * sizeof( ObjectPlane ) ) );
//	CUDA_CHECK_RETURN( hipMemcpy( objPlane_device, &plane, sizeof( ObjectPlane ), hipMemcpyHostToDevice ) );
	//CUDA_CHECK_RETURN( hipMemset( image_device, -1, image_size * sizeof( Color ) ) );

	uint32_t gridSize = image_size / 256;
	printf( "%u\n", gridSize );
	calculate_light<<< gridSize, 256 >>>( image_device, image.width, image.height, cameraPos, viewport, planesNumber + spheresNumber );


	CUDA_CHECK_RETURN( hipDeviceSynchronize() );
	CUDA_CHECK_RETURN( hipGetLastError() );

	CUDA_CHECK_RETURN( hipMemcpy( image.image, image_device, image_size * sizeof( Color ), hipMemcpyDeviceToHost ) );

	CUDA_CHECK_RETURN( hipFree( ( void* )image_device ) );
	//CUDA_CHECK_RETURN( hipFree( ( void* )objPlane_device ) );
	CUDA_CHECK_RETURN( hipDeviceReset() );

	save_png( "out.png", image );

	return 0;
}

/*
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include "cuda.hpp"

#define EPSILON 0.001

HOST_DEVICE inline float4 operator+( const float4 & a, const float4 & b )
{
	return make_float4( a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w );
}

HOST_DEVICE inline float4 operator-( const float4 & a, const float4 & b )
{
	return make_float4( a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w );
}

HOST_DEVICE inline float4 operator*( const float4 & a, const float4 & b )
{
	return make_float4( a.y * b.z - a.z * b.y,
		  	  	  	    a.z * b.x - a.x * b.z,
		  	  	  	    a.x * b.y - a.y * b.x,
		  	  	  	    0.0f );
}

HOST_DEVICE inline float4 operator*( const float4 & a, const float & b )
{
	return make_float4( a.x * b, a.y * b, a.z * b, a.w * b );
}

HOST_DEVICE inline float dot( const float4 & a, const float4 & b )
{
	return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
}

HOST_DEVICE inline float length( const float4 & a )
{
	return sqrtf( a.x * a.x + a.y * a.y + a.z * a.z + a.w * a.w );
}

HOST_DEVICE inline void normalize( float4 & v )
{
	float l = length( v );
	v.x = v.x / l;
	v.y = v.y / l;
	v.z = v.z / l;
	v.w = v.w / l;
}

HOST_DEVICE inline float4 reflect( const float4 & v, const float4 & normal )
{
	return v - normal * 2.0f * dot( v, normal ) ;
}

struct Ray
{
	float4 		m_start_point;
	float4 		m_vector;
	HOST_DEVICE Ray(){}
	HOST_DEVICE Ray( const float4 & end_point, const float4 & start_point )
		: m_vector( end_point - start_point ),
		  m_start_point( start_point )
	{
		normalize( m_vector );
	}
	HOST_DEVICE void operator()( const float4 & end_point, const float4 & start_point )
	{
		m_vector = end_point - start_point;
		m_start_point = start_point;
		normalize( m_vector );
	}
	HOST_DEVICE float4 point( const float & t ) const
	{
		const float & x0 = m_start_point.x;
		const float & y0 = m_start_point.y;
		const float & z0 = m_start_point.z;
		const float & w0 = m_start_point.w;
		const float & alfa  = m_vector.x;
		const float & beta  = m_vector.y;
		const float & gamma = m_vector.z;
		const float & delta = m_start_point.w;
		return make_float4( x0 + alfa * t,
				            y0 + beta * t,
			                z0 + gamma * t,
			                w0 + delta * t );
	}
	HOST_DEVICE ~Ray(){}
} __attribute__ ( ( aligned( 16 ) ) );


struct Intersection
{
    float4  	point;
    int     	at_object;
    float4  	normal;
    Ray 	    reflect_ray;
    Ray 		refract_ray;
    float   	reflect_amount;
    float   	pixel;
} __attribute__ ( ( aligned( 16 ) ) );


struct Material
{
	float4 		m_ambient;
	float4 		m_diffuse;
	float4 		m_specular;
	float 		m_beta;
	float 		m_phong;
	float 		m_refract_amount;
	float 		m_refract_coef;
} __attribute__ ( ( aligned( 16 ) ) );

/*
struct Object
{
	Material 	m_material;
	virtual ~Object() {}
	void get_reflect_refract_rays( const Ray & ray, Intersection & intersection )
	{
		float4 i = ray.m_vector;
		float4 n = intersection.normal;

		intersection.reflect_ray.m_vector = reflect( i, n );
		normalize( intersection.reflect_ray.m_vector );
		intersection.reflect_ray.m_start_point = intersection.point;
		intersection.reflect_amount = 1.0f;

		if( m_material.m_refract_amount > 0.0f ){
			float refract_coef = m_material.m_refract_coef;
			float cos_i = -dot( i, n );
			if( cos_i < 0.0f ){
				n = n * -1.0f;
				cos_i = -dot( i, n );
				refract_coef = 1.0f / refract_coef;
			}
			float sin2_t = refract_coef * refract_coef * ( 1.0f - cos_i*cos_i );
			if( sin2_t <= 1.0f ){
				float cos_t = sqrt( 1.0f - sin2_t );
				intersection.refract_ray.m_vector = i * refract_coef  + n * ( refract_coef * cos_i - cos_t );
				normalize( intersection.refract_ray.m_vector );
				intersection.refract_ray.m_start_point = intersection.point;
				float Rorto  = ( cos_i - refract_coef * cos_t ) / ( cos_i + refract_coef * cos_t );
				float Rparal = ( refract_coef * cos_i - cos_t ) / ( refract_coef * cos_i + cos_t );
				intersection.reflect_amount = ( Rorto * Rorto + Rparal * Rparal ) / 2.0f;
			}
			else{
				intersection.refract_ray = intersection.reflect_ray;
			}
		}

	}
	bool is_in_border( const float & v, const float & b1, const float & b2, float & t ){
		t = ( v - b1 ) / ( b2 - b1 );
		t = t > 1.0f ? 1.0f : t;
		if( v >= b1 && v <= b2 )// || ( v >= b2 && v <= b1 ) )
				return true;
		if( fabs( v - b1 ) < EPSILON || fabs( v - b2 ) < EPSILON )
			return true;
		return false;
	}
	virtual bool RayIntersect( const Ray & ray, Intersection & intersection ) = 0;
};
*/

